#include <iostream>
#include <vector>
#include <thread>
#include <mutex>
#include <chrono>
#include <math.h>

#include "hip/hip_runtime.h"

#include "hip/hip_runtime_api.h"
#include "hip/device_functions.h"						// synchronization


using namespace std;



/**
 * @brief Struct to store prime factors and their exponents
 */
struct factor_exponent {
    unsigned long long factor;
    int exponent;
};


#define MAX_PRIMES 100

// Global array statically allocated in the device memory
__device__ factor_exponent d_primes[MAX_PRIMES];
// Global variable to store the number of prime factors found
__device__ unsigned int d_primes_count = 0;


/**
 * @brief Function to check if a number is prime
 *
 * @param n number to check if prime
 * @return true if prime, false otherwise
 */
__device__ bool isPrime(unsigned long long n) {
    if (n <= 1) return false;
    if (n <= 3) return true;
    if (n % 3 == 0) return false;
    for (unsigned long long i = 5; i * i <= n; i += 6) {
        if (n % i == 0 || n % (i + 2) == 0) return false;
    }
    return true;
}


/**
 * @brief Trial division function to find prime factors in a range
 *
 * @param start integer to start from
 * @param end integer to end at
 * @param num number to find prime factors of
 * @param primes vector to store prime factors
 */
__global__ void findPrimesInRange(unsigned long long *d_start, unsigned long long *d_end, unsigned long long num) {

    // (START) DEBUG
    // {
    //     lock_guard<mutex> lock(mtx);
    //     // Get the thread id
    //     thread::id this_id = this_thread::get_id();
    //     cout << "#START: Thread ID: " << this_id << " is running on core: " << GetCurrentProcessorNumber() << endl;
    // }
    // (END) DEBUG

    // Get the blockIdx.x
    unsigned long long j = blockIdx.x;


    // For avoiding problems of divergence, 
    // just execute the first thread of each warp
    // (in the considered architecture, the warp size is 32)
    if(threadIdx.x % 32 == 0){

        // Print the range of the current block
        // printf("Block ID: %d - Range: %llu - %llu\n", j, d_start[j], d_end[j]);

        // check all numbers in the range
        for (unsigned long long i = d_start[j]; i <= d_end[j]; i += 2) {

            if ((num % i) == 0) {

                // continue dividing as long as possible
                // this way we avoid adding the same factor multiple times
                int exponent = 0;
                while (num % i == 0) {
                    exponent++;
                    num /= i;
                }

                // 
                if (isPrime(i)) {
                    int index = atomicAdd(&d_primes_count, 1); // atomicAdd returns the old value
                    d_primes[index].factor = i; 
                    d_primes[index].exponent = exponent;
                }
            }
        }

    }

    


    // (START) DEBUG
    // {
    //     lock_guard<mutex> lock(mtx);
    //     // Get the thread id
    //     thread::id this_id = this_thread::get_id();
    //     cout << "#END: Thread ID: " << this_id << " is running on core: " << GetCurrentProcessorNumber() << endl;
    // }
    // (END) DEBUG

}


/**
 * @brief Main function for parallel factorization, using trial division algorithm
 *
 * @param num number to find prime factors of
 * @param numThreads number of threads to use
 * @return vector<factor_exponent> vector of prime factors
 */
vector<factor_exponent> CUDA_TrialDivision(unsigned long long num, int num_CUDA_blocks) {

    // vector to store the prime factors
    vector<factor_exponent> primes;

    // store the original number for later use
    unsigned long long old_num = num;

    // checking in advance if the number is divisible by 2
    // to avoid checking even numbers afterwards
    if (num % 2 == 0) {
        int exponent = 0;
        while (num % 2 == 0) {
            exponent++;
            num /= 2;
        }
        primes.push_back({ 2, exponent });
    }

    // now the interval to check is nearly halved
    // as checking divisibility by even numbers 
    // is not needed anymore

    unsigned long long sqrt_num = (unsigned long long) sqrt(num);

    // divide the work equally among the different CUDA Blocks
    unsigned long long range = sqrt_num / num_CUDA_blocks;

    unsigned long long start[num_CUDA_blocks];
    unsigned long long end[num_CUDA_blocks];

    // define the start and end of the range for the first Block
    start[0] = 3;
    end[0] = (range % 2 == 0) ? range + 1 : range;

    // define the start and end of the range for the other Blocks
    for (int i = 1; i < num_CUDA_blocks; i++) {
        start[i] = end[i - 1] + 2;
        if (range % 2 == 0) {
            end[i] = start[i] + range;
        }
        else {
            end[i] = start[i] + range - 1;
        }
    }
    end[num_CUDA_blocks - 1] = sqrt_num;

    // Creating the device counterparts of the start and end arrays
    unsigned long long* d_start;
    unsigned long long* d_end;

    hipMalloc((void **)&d_start, sizeof(unsigned long long) * num_CUDA_blocks);
    hipMalloc((void **)&d_end, sizeof(unsigned long long) * num_CUDA_blocks);

    hipMemcpy(d_start, start, sizeof(unsigned long long) * num_CUDA_blocks, hipMemcpyHostToDevice);
    hipMemcpy(d_end, end, sizeof(unsigned long long) * num_CUDA_blocks, hipMemcpyHostToDevice);

    dim3 blocksPerGrid(num_CUDA_blocks, 1, 1);
    dim3 threadsPerBlock(32, 1, 1);

    float elapsedTime;
    hipEvent_t time_start, time_stop;
    
    hipEventCreate(&time_start);
    hipEventCreate(&time_stop);
    hipEventRecord(time_start, 0);

    // call the kernel function to find the prime factors of the number
    findPrimesInRange<<<blocksPerGrid, threadsPerBlock>>>(d_start, d_end, num);

    hipEventRecord(time_stop, 0);
    hipEventSynchronize(time_stop);
    hipEventElapsedTime(&elapsedTime, time_start, time_stop);

    hipEventDestroy(time_start);
    hipEventDestroy(time_stop);

    // DEBUG (START)
    cout<<"Time taken by the GPU: "<<elapsedTime<<" milliseconds."<<endl;
    // DEBUG (END)

    
    unsigned int host_primes_count;
    hipMemcpyFromSymbol(&host_primes_count, HIP_SYMBOL(d_primes_count), sizeof(unsigned int));

    factor_exponent* h_primes = new factor_exponent[host_primes_count];
    hipMemcpyFromSymbol(h_primes, HIP_SYMBOL(d_primes), sizeof(factor_exponent) * host_primes_count);

    for (int i = 0; i < host_primes_count; i++) {
        primes.push_back(h_primes[i]);
    }

    delete[] h_primes;
    hipFree(d_start);
    hipFree(d_end);

    // if primes is empty than the number is prime so add it to the vector
    if (primes.empty()) {
        primes.push_back({ num, 1 });
    }
    else {

        // check if all the factors have been found 
        // (otherwise a prime factor larger than the 
        // square root of the number is missing)

        // add the (possible) missing prime factor
        unsigned long long product = 1;
        for (vector<factor_exponent>::iterator it = primes.begin(); it != primes.end(); ++it) {

            // calculate the product of the prime factors by multiplying each 
            // factor by itself for the number of times stated by the exponent
            // not using pow() as it cuts off large numbers
            for (int i = 0; i < it->exponent; i++) {
                product *= it->factor;
            }

        }
        // if the product is different from the original number,
        // then the missing prime factor is the number divided by the product
        if (product != old_num) {
            primes.push_back({ old_num / product, 1 });
        }
    }

    return primes;
}


int main(int argc, char* argv[]) {

    // check if the number of arguments is correct
    if (argc != 4) {
        cout << "Please provide:\n1) Number of CUDA Blocks\n2) Number to be Factorized\n3) 0 or 1 to execute the program in BASH or USER mode." << endl;
        return 1;
    }

    // get the number of CUDA blocks from the command line argument
    int NUM_CUDA_BLOCKS = atoi(argv[1]);

    // get the number from the command line argument
    // and convert it to unsigned long long using stoull
    unsigned long long NUMBER = stoull(argv[2]);

    // get the mode (0: bash, 1: user)
    bool EXECUTION_MODE = atoi(argv[3]);

    // start measuring time (HOST point of view)
    chrono::steady_clock::time_point start = chrono::steady_clock::now();

    // find the prime factors of the number 
    vector<factor_exponent> prime_factors = CUDA_TrialDivision(NUMBER, NUM_CUDA_BLOCKS);

    // stop measuring time
    chrono::steady_clock::time_point end = chrono::steady_clock::now();

    // calculate the time duration
    chrono::milliseconds duration = chrono::duration_cast<chrono::milliseconds>(end - start);

    // depending on the execution mode, print some informations on screen
    if (EXECUTION_MODE) {
        cout << "Time taken: " << duration.count() << " milliseconds." << endl;

        cout << NUMBER << " = ";
        for (vector<factor_exponent>::iterator it = prime_factors.begin(); it != prime_factors.end(); ++it) {
            cout << it->factor << "^" << it->exponent;
            // print a * between prime factors except for the last one
            if (next(it) != prime_factors.end()) {
                cout << " * ";
            }
        }
        cout << endl;
    }
    else {
        cout << duration.count() << endl;
    }

    return 0;
}